#include "hip/hip_runtime.h"
#include <cstdlib>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/tuple.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>

//Required constants.
#define EPSILON 1E-4 
#define MAXITER 7000



//Useful functions for finding partial derivatives and exact solution(assumption for existence of the solution).
float exact(float x,float y,float z)
{
  return(x*x+y*y+z*z) ; 
}
//Partial derivation function with respect to uxx, uyy and uzz.
float f(float x,float y,float z) 
{
  return(6.0) ; 
}

/*
Saxpy functor to solve linear differential equation.
*/
struct saxpy_functor: public thrust::binary_function<float,float,float>
{
	 int prev;
	 int cur;
	const int N;
	float * ptr;
  //Constructor takes previous index, current index, N and a vector which holds all points.
	saxpy_functor(int _prev,int _cur,int _N,float *_ptr):prev(_prev),cur(_cur),N(_N),ptr(_ptr){}

    //Jacobi iteration is done here.
	__host__ __device__
		float operator()( const thrust::tuple<int,int,int> &y,const float &x) const{
			int a,b,c;
			a=thrust::get<0>(y);
			b=thrust::get<1>(y);
			c=thrust::get<2>(y);
			float h=1.0/(N-1);
			ptr[2*(a*N*N+b*N+c)+cur]=(1.0/6)*(ptr[2*((a-1)*N*N+b*N+c)+prev]+ptr[2*((a+1)*N*N+b*N+c)+prev]+ptr[2*(a*N*N+(b+1)*N+c)+prev]+ptr[2*(a*N*N+(b-1)*N+c)+prev]+ptr[2*(a*N*N+b*N+c+1)+prev]+ptr[2*(a*N*N+b*N+c-1)+prev])-(h*h);
  			float diff=ptr[2*(a*N*N+b*N+c)+cur]-ptr[2*(a*N*N+b*N+c)+prev];
			return diff*diff;
	}

};
 int i,j,k ; 
using namespace std;


int main(int argc, char**argv)
{
  float h,x,y,z ; 
  if(argc!=2){
  	cout<<"Usage: ./executable N";
  	exit(-1);
  }
  
  int N;
  N=atoi(argv[1]);
  h = 1.0/N ; 
  //Vector initialization to hold all numbers. It will be a 3D vector. (It will be flattened later.)
  //These variables are multiplied by 2 because flattened vector will hold both previous and current values.
  unsigned long long int start_index=2*((N+1)*(N+1) + (N+1) + 1);
  unsigned long long int end_index=2*((N+1)*(N+1)*(N-1) + (N+1)*(N-1) + (N-1));

  int dims[4]={N+1,N+1,N+1,2};
  
  //Total dimension required for 1D array.
  unsigned long long int total_dims=dims[0]*dims[1]*dims[2]*dims[3];
  //Holds all numbers. Flattened version of 3D vector.
  thrust::host_vector <float> nums(total_dims);

  // Assigning non-boundary points for all 3 dimensions.
  for(i=start_index;i<=end_index;i++){
  	nums[i]=0.0;
  }

  // Assigning boundary points for dimension X.
  for(i=0 ; i <= N ; i++) {
    x = i*h ;
    nums[2*(i*dims[0]*dims[1])]=exact(x,0.0,0.0);
    nums[2*(i*dims[0]*dims[1]+N*dims[0]+N)]=exact(x,1.0,1.0);
  }
  // Assigning boundary points for dimension Y.
  for(j=0 ; j <= N ; j++) {
    y = j*h ; 
     nums[2*j*dims[0]] =exact(0.0,y,0.0);
    nums[2*(N*dims[0]*dims[1]+j*dims[0]+N)]=exact(1.0,y,1.0);

}
// Assigning boundary points for dimension Z.
 for(k=0 ; k <= N ; k++) {
 	z=k*h;
     nums[2*k] =exact(0.0,0.0,z);
    nums[2*(N*dims[0]*dims[1]+N*dims[0]+k)]=exact(1.0,1.0,z);     
}
	
	//Transfer from host to device vector.
	thrust::device_vector <float> Dnums=nums;
	thrust::device_vector <float> result((N-1)*(N-1)*(N-1));
	thrust::device_vector<thrust::tuple<float,float,float> > iterations;
	for(i=1 ; i < N ; i++) {
      for(j=1 ; j < N ; j++) {
        for(k=1;k<N;k++){
        	iterations.push_back(thrust::make_tuple(i,j,k));
        }
    }
}

	

	// iteration loop until convergence.
	
 unsigned int iter,prev,cur ; 
 float sum;
 iter = 0 ; 
  prev=0;
  sum = 1.0E30 ; 
  //Start time of parallel execution
const clock_t begin_time = clock();
//Solves differential equation until sum is greater than epsilon
  while( (sum > EPSILON)   && (iter < MAXITER) ) {
     cur = (prev + 1) % 2 ; 
   	sum=0.0;
    //Transformation is required to solve linear system. Saxpy does Jacobi iteration.
   	thrust::transform(iterations.begin(),iterations.end(),Dnums.begin(),result.begin(),saxpy_functor(prev,cur,N+1,thrust::raw_pointer_cast(&(Dnums[0]))));
  	float tot=thrust::reduce(result.begin(),result.end(),(float)0,thrust::plus<float>());
    //Result is reduced and added.
    sum+=tot;
    iter = iter + 1 ;
    prev = cur ; 
   // printf("%f\n",sum);
  }
//End time of parallel execution
  printf("%d\t%d\t%.5f\n",N,iter,float( clock () - begin_time ) /  CLOCKS_PER_SEC);
  	
  	
   
	
 	

 
  return 0;
}